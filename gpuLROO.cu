#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>   
#include <string>
#include <vector>

#include "myLROO.h"

using namespace std;

__device__ const double pi[3][7] = {{0.21484375, 0.3671875, 0.23046875, 0.1875},
                         {0.1174035788, 0.242955959, 0.249363483,
                          0.17517706, 0.102701071, 0.112398847},
                         {0.0882, 0.2092, 0.2483, 0.1933, 0.1208,
                          0.0675, 0.0727}};

__device__ const int v[3][7] = {{1, 2, 3, 4}, {4, 5, 6, 7, 8, 9}, 
                     {10, 11, 12, 13, 14, 15, 16}};

__device__ const int M_c[3] = {8, 128, 10000};
__device__ const int K_c[3] = {3, 5, 6};

__device__ int gpu_specify_type(int n){
    int type;
    if (n < 128){
        return -1;
    }
    else if (n < 6272){
        type = 0;
    }
    else if (n < 750000){
        type = 1;
    }
    else {
        type = 2;
    }
    return type;
}

__device__ double gpuLongestRunOfOnes(int n, int data[]){
    int K, M, N,type;
    double v_measured[7] = {0}, X = 0; 
    type = gpu_specify_type(n);
    M = M_c[type];
    K = K_c[type];
    N = n/M;
    int run, longest_run;
    for (int i=0; i<N; i++){
        run = 0;
        longest_run = 0;
        for (int j=0; j<M; j++){
            if (data[i*M+j] == 1){
                run++;
            }
            if (longest_run < run){
                longest_run = run;
            }
            if (data[i*M+j] == 0){
                run = 0;
            }
        }
        if (longest_run<=v[type][0]){
            v_measured[0]++;
        }
        else if(longest_run>=v[type][K]){
            v_measured[K]++;
        }
        else{
            for (int j=1; j<K; j++){
                if( v[type][j] == longest_run){
                    v_measured[j]++;
                    break;
                }
            }
        }
    }
    int sum_control = 0;
    for (int i=0; i<K+1; i++){
        sum_control += v_measured[i];
    }
    if (sum_control != N) return -1;
    for (int i=0; i<K+1; i++){
        X+= pow(v_measured[i] - N*pi[type][i], 2)/(N*pi[type][i]);
    }
    return X;
}

__global__ void gpu(int *a, double *b, int z, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<z) b[i] = gpuLongestRunOfOnes(n, &a[i*n]);
}
