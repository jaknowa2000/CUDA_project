#include "hip/hip_runtime.h"
#include <iostream>   
#include <vector>
#include <stdlib.h>
#include <string>
#include <memory>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cmath>

#include "files_service.h"
#include "gpuLROO.h"
#include "myLROO.h"
#include "cuda_service.h"

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>

using namespace std;    

const string path = "../NIST-Statistical-Test-Suite/sts/data3/";
const string extension = ".txt";
        
vector<double> cuda_service(){
    hipError_t what;
    int n;
    vector<string> list_names = file_names(path, extension);
    vector<vector<char>> list_of_data = read_all_files(list_names, path);
    vector<char> data_a;
    size_t bytes = 0;
    for (int i=0; i<list_of_data.size(); i++){
        data_a.insert(data_a.begin() + i*list_of_data[i].size(), list_of_data[i].begin(), list_of_data[i].end());  
        bytes = bytes + list_of_data[i].size() * sizeof(char);
    }
    int n_files = list_of_data.size();
    n = list_of_data[0].size();
    char *d_a;
    int THREADS = 256;
    double *d_b;
    what = hipMalloc(&d_a, bytes);
    what = hipMalloc(&d_b, n_files * sizeof(double));
    what = hipMemcpy(d_a, data_a.data(), bytes, hipMemcpyHostToDevice);
    int BLOCKS = (n_files + THREADS - 1)/THREADS;
    hipEvent_t start , stop ; 
    what = hipEventCreate (& start ); 
    what = hipEventCreate (& stop );
    what = hipEventRecord(start);  
    gpu<<<BLOCKS, THREADS>>>(d_a, d_b, n_files, n);
    what = hipEventRecord(stop);
    vector<double> result(n_files * sizeof(double));
    what = hipMemcpy(result.data(), d_b, n_files * sizeof(double), hipMemcpyDeviceToHost);
    what = hipDeviceSynchronize();
    what = hipEventSynchronize(stop);
    float milliseconds = 0;
    what = hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"WHAT!: "<<what<<endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "Czas testow na GPU: "<<milliseconds<<" ms" << endl;
    hipFree(d_a);
    hipFree(d_b);
    return result;
}

vector<double> cpu_service(){
    vector<string> list_names = file_names(path, extension);
    vector<vector<char>> list_of_data = read_all_files(list_names, path);
    vector<double> result;
    auto start = chrono::steady_clock::now();
    for (int i=0; i<list_of_data.size(); i++){
        result.push_back(myLongestRunOfOnes(list_of_data[i].size(), list_of_data[i]));
    }
    auto end = chrono::steady_clock::now();
    cout << "Czas testow na CPU: "<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms" << endl;
    return result;
}

void comparison(){
    cout<<"----------------------------------------"<<endl;
    vector<double> result1 = cuda_service();
    vector<double> result2 = cpu_service();
    cout<<endl<<"----------------------------------------"<<endl;
    int test_passed=0, test_failed=0;
    for(int i=0; i<result2.size(); i++){
        if(fabs(result1[i] - result2[i]) < 0.000001) test_passed++;
        else test_failed++;
    }
    cout<<endl<<"TEST PASSED: "<<test_passed<<endl;
    cout<<"TEST FAILED: "<<test_failed<<endl<<endl;
    cout<<"----------------------------------------"<<endl;
}
