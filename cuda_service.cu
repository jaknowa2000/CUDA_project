#include "hip/hip_runtime.h"
#include <iostream>   
#include <vector>
#include <string>
#include <chrono>
#include <cmath>

#include "hip/hip_runtime_api.h"

#include "files_service.h"
#include "gpu_LROO.h"
#include "my_LROO.h"
#include "cuda_service.h"


using namespace std;    

const string path = "../NIST-Statistical-Test-Suite/sts/data/";
const string extension = ".txt";
        
vector<double> cuda_service(){
    int n, n_files, THREADS = 256, BLOCKS;
    char *d_a;
    double *d_b;
    float milliseconds = 0;
    vector<string> list_names = file_names(path, extension);
    vector<vector<char>> list_of_data = read_all_files(list_names, path);
    vector<char> data_a;
    size_t bytes = 0;
    hipEvent_t start , stop ; 

    for (int i=0; i<list_of_data.size(); i++){
        data_a.insert(data_a.begin() + i*list_of_data[i].size(), list_of_data[i].begin(), list_of_data[i].end());  
        bytes = bytes + list_of_data[i].size() * sizeof(char);
    }
    n_files = list_of_data.size();
    n = list_of_data[0].size();
    BLOCKS = (n_files + THREADS - 1)/THREADS;
    vector<double> result(n_files * sizeof(double));
    checkCudaErrors(hipMalloc(&d_a, bytes));
    checkCudaErrors(hipMalloc(&d_b, n_files * sizeof(double)));
    checkCudaErrors(hipMemcpy(d_a, data_a.data(), bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventCreate (& start )); 
    checkCudaErrors(hipEventCreate (& stop ));
    checkCudaErrors(hipEventRecord(start));  
    gpu<<<BLOCKS, THREADS>>>(d_a, d_b, n_files, n);
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipMemcpy(result.data(), d_b, n_files * sizeof(double), hipMemcpyDeviceToHost));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    cout << "Time of tests on the GPU: "<<milliseconds<<" ms" << endl;
    hipFree(d_a);
    hipFree(d_b);
    return result;
}

vector<double> cpu_service(){
    vector<string> list_names = file_names(path, extension);
    vector<vector<char>> list_of_data = read_all_files(list_names, path);
    vector<double> result;
    auto start = chrono::steady_clock::now();
    for (int i=0; i<list_of_data.size(); i++){
        result.push_back(myLongestRunOfOnes(list_of_data[i].size(), list_of_data[i]));
    }
    auto end = chrono::steady_clock::now();
    cout << "Time of tests on the CPU: "<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms" << endl;
    return result;
}

void comparison(){
    cout<<"----------------------------------------"<<endl;
    vector<double> result1 = cuda_service();
    vector<double> result2 = cpu_service();
    cout<<"----------------------------------------"<<endl<<endl;
    cout<<"----------------------------------------";
    int test_passed=0, test_failed=0;
    for(int i=0; i<result2.size(); i++){
        if(fabs(result1[i] - result2[i]) < 0.000001) test_passed++;
        else test_failed++;
    }
    cout<<endl<<"TEST PASSED: "<<test_passed<<endl;
    cout<<"TEST FAILED: "<<test_failed<<endl;
    cout<<"----------------------------------------"<<endl<<endl;
}
