#include "hip/hip_runtime.h"
#include <iostream>   
#include <vector>
#include <stdlib.h>
#include <string>
#include <memory>
#include <chrono>
#include <hip/hip_runtime.h>
#include <cmath>

#include "files_service.h"
#include "gpuLROO.h"
#include "myLROO.h"
#include "cuda_service.h"

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>

using namespace std;    

const string path = "../NIST-Statistical-Test-Suite/sts/data3/";
const string extension = ".txt";
        
vector<double> cuda_service(){
    hipError_t what;
    int n;
    vector<string> list_names = file_names(path, extension);
    vector<vector<uint8_t>> list_of_data = read_all_files(list_names, path);
    vector<uint8_t> data_a;
    size_t bytes = 0;
    for (int i=0; i<list_of_data.size(); i++){
        data_a.insert(data_a.begin() + i*list_of_data[i].size(), list_of_data[i].begin(), list_of_data[i].end());  
        bytes = bytes + list_of_data[i].size() * sizeof(uint8_t);
    }
    cout<<"DATA A SIZE: "<<data_a.size()<<endl;
    int n_files = list_of_data.size();
    n = list_of_data[0].size();
    cout<<"N: "<<n<<endl;
    uint8_t *d_a;
    int THREADS = 256;
    double *d_b;
    cout<<"BYTES: "<<bytes<<endl;
    what = hipMalloc(&d_a, bytes);
    cout<<"WHAT1: "<<what<<endl;
    what = hipMalloc(&d_b, n_files * sizeof(double));
    cout<<"WHAT2: "<<what<<endl;
    what = hipMemcpy(d_a, data_a.data(), bytes, hipMemcpyHostToDevice);
    cout<<"WHAT3: "<<what<<endl;
    int BLOCKS = (n_files + THREADS - 1)/THREADS;
    hipEvent_t start , stop ; 
    what = hipEventCreate (& start ); 
    cout<<"WHAT4: "<<what<<endl;
    what = hipEventCreate (& stop );
    cout<<"WHAT5: "<<what<<endl;
    what = hipEventRecord(start);  
    cout<<"WHAT6: "<<what<<endl;
    gpu<<<BLOCKS, THREADS>>>(d_a, d_b, n_files, n);
    what = hipEventRecord(stop);
    cout<<"WHAT7: "<<what<<endl;
    vector<double> result(n_files * sizeof(double));
    cout<<"RESULTSIZE: "<<result.size()<<endl;
    what = hipMemcpy(result.data(), d_b, n_files * sizeof(double), hipMemcpyDeviceToHost);
    cout<<"WHAT8: "<<what<<endl;
    what = hipDeviceSynchronize();
    cout<<"WHAT9: "<<what<<endl;
    what = hipEventSynchronize(stop);
    cout<<"WHAT10: "<<what<<endl;
    float milliseconds = 0;
    what = hipEventElapsedTime(&milliseconds, start, stop);
    cout<<"WHAT!: "<<what;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "Czas testow na GPU: "<<milliseconds<<" ms" << endl;
    hipFree(d_a);
    hipFree(d_b);
    return result;
}

vector<double> cpu_service(){
    vector<string> list_names = file_names(path, extension);
    vector<vector<uint8_t>> list_of_data = read_all_files(list_names, path);
    vector<double> result;
    auto start = chrono::steady_clock::now();
    for (int i=0; i<list_of_data.size(); i++){
        result.push_back(myLongestRunOfOnes(list_of_data[i].size(), list_of_data[i]));
    }
    auto end = chrono::steady_clock::now();
    cout << "Czas testow na CPU: "<< chrono::duration_cast<chrono::milliseconds>(end - start).count()
         << " ms" << endl;
    return result;
}

void comparison(){
    vector<double> result1 = cuda_service();
    vector<double> result2 = cpu_service();
    int test_passed=0, test_failed=0;
    for(int i=0; i<result2.size(); i++){
        if(fabs(result1[i] - result2[i]) < 0.01) test_passed++;
        else test_failed++;
    }
    cout<<"RESULT1: "<<result1[99999]<<endl;
    cout<<"RESULT2: "<<result2[99999]<<endl;
    cout<<"RESULT1: "<<result1[999999]<<endl;
    cout<<"RESULT2: "<<result2[999999]<<endl;
    cout<<endl<<"TEST PASSED: "<<test_passed<<endl;
    cout<<"TEST FAILED: "<<test_failed<<endl<<endl;

}
