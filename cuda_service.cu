#include <iostream>   
#include <vector>
#include <stdlib.h>
#include <string>
#include <memory>

#include "files_service.h"
#include "gpuLROO.h"
#include "cuda_service.h"

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>

using namespace std;      
        
vector<double> cuda_service(){
    int n;
    string path = "../../data/";
    string extension = ".txt";
    vector<string> list_names = file_names(path, extension);
    //for (int i=0; i<list_names.size(); i++){
        //cout<<list_names[i]<<endl;
    //}
    vector<vector<int>> list_of_data = read_all_files(list_names, path);
    vector<int> data_a;
    size_t bytes = 0;
    for (int i=0; i<list_of_data.size(); i++){
        data_a.insert(data_a.begin() + i*list_of_data[i].size(), list_of_data[i].begin(), list_of_data[i].end());  
        bytes = bytes + list_of_data[i].size() * sizeof(int);
    }
    int n_files = list_of_data.size();
    n = list_of_data[0].size();
    int *d_a;
    int THREADS = 256;
    double *d_b;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, n_files * sizeof(double));
    hipMemcpy(d_a, data_a.data(), bytes, hipMemcpyHostToDevice);
    int BLOCKS = (n_files + THREADS - 1)/THREADS;
    gpu<<<BLOCKS, THREADS>>>(d_a, d_b, n_files, n);
    vector<double> result(n_files * sizeof(double));
    hipMemcpy(result.data(), d_b, n_files * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    return result;
}